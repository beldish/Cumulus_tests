#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
       C[idx] = A[idx] + B[idx];
       }
}

int main() {
    int N = 1000000; // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
    h_A[i] = i;
    h_B[i] = i * 2;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data from host to device (CPU -> GPU)
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Set up execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vector addition kernel on the GPU
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy the result from device to host (GPU -> CPU)
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < 5; i++) {
    std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
    }
    